//#include "kernel.h"
#include "bboxUtils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "plugin.h"
#include<NvInferPlugin.h>
#include<NvInferPluginUtils.h>
#include<NvInfer.h>
#include<NvInferRuntime.h>

using namespace nvinfer1;
using namespace nvinfer1::plugin;

pluginStatus_t sortScoresPerImage(
	hipStream_t stream,
	const int num_images,
	const int num_items_per_image,
	const DataType DT_SCORE,
	void* unsorted_scores,
	void* unsorted_bbox_indices,
	void* sorted_scores,
	void* sorted_bbox_indices,
	void* workspace);

pluginStatus_t sortScoresPerClass(
	hipStream_t stream,
	const int num,
	const int num_classes,
	const int num_preds_per_class,
	const int background_label_id,
	const float confidence_threshold,
	const DataType DT_SCORE,
	void* conf_scores_gpu,
	void* index_array_gpu,
	void* workspace);

pluginStatus_t decodeBBoxes(
	hipStream_t stream,
	const int nthreads,
	const CodeTypeSSD code_type,
	const bool variance_encoded_in_target,
	const int num_priors,
	const bool share_location,
	const int num_loc_classes,
	const int background_label_id,
	const bool clip_bbox,
	const DataType DT_BBOX,
	const void* loc_data,
	const void* prior_data,
	void* bbox_data);

pluginStatus_t permuteData(hipStream_t stream,
	const int nthreads,
	const int num_classes,
	const int num_data,
	const int num_dim,
	const DataType DT_DATA,
	bool confSigmoid,
	const void* data,
	void* new_data);

pluginStatus_t gatherTopDetections(
	hipStream_t stream,
	const bool shareLocation,
	const int numImages,
	const int numPredsPerClass,
	const int numClasses,
	const int topK,
	const int keepTopK,
	const DataType DT_BBOX,
	const DataType DT_SCORE,
	const void* indices,
	const void* scores,
	const void* bboxData,
	void* keepCount,
	void* topDetections);

pluginStatus_t allClassNMS_m(hipStream_t stream,
	const int num,
	const int num_classes,
	const int num_preds_per_class,
	const int top_k,
	const float nms_threshold,
	const bool share_location,
	const bool isNormalized,
	const DataType DT_SCORE,
	const DataType DT_BBOX,
	void* bbox_data,
	void* beforeNMS_scores,
	void* beforeNMS_index_array,
	void* afterNMS_scores,
	void* afterNMS_index_array,
	bool flipXY);

size_t detectionForwardBBoxDataSize(int N, int C1, DataType DT_BBOX)
{
	if (DT_BBOX == DataType::kFLOAT)
	{
		return N * C1 * sizeof(float);
	}

	printf("Only FP32 type bounding boxes are supported.\n");
	return (size_t)-1;
}

size_t detectionForwardBBoxPermuteSize(bool shareLocation, int N, int C1, DataType DT_BBOX)
{
	if (DT_BBOX == DataType::kFLOAT)
	{
		return shareLocation ? 0 : N * C1 * sizeof(float);
	}
	printf("Only FP32 type bounding boxes are supported.\n");
	return (size_t)-1;
}

size_t detectionForwardPreNMSSize(int N, int C2)
{
	ASSERT(sizeof(float) == sizeof(int));
	return N * C2 * sizeof(float);
}

size_t detectionForwardPostNMSSize(int N, int numClasses, int topK)
{
	ASSERT(sizeof(float) == sizeof(int));
	return N * numClasses * topK * sizeof(float);
}


pluginStatus_t detectionInference(
	hipStream_t stream,
	const int N,
	const int C1,
	const int C2,
	const bool shareLocation,
	const bool varianceEncodedInTarget,
	const int backgroundLabelId,
	const int numPredsPerClass,
	const int numClasses,
	const int topK,
	const int keepTopK,
	const float confidenceThreshold,
	const float nmsThreshold,
	const CodeTypeSSD codeType,
	const DataType DT_BBOX,
	const void* locData,
	const void* priorData,
	const DataType DT_SCORE,
	const void* confData,
	void* keepCount,
	void* topDetections,
	void* workspace,
	bool isNormalized,
	bool confSigmoid)
{
	// Batch size * number bbox per sample * 4 = total number of bounding boxes * 4
	const int locCount = N * C1;
	// Do not clip the bounding box that goes outside the image
	const bool clipBBox = false;
	/*
	 * shareLocation
	 * Bounding box are shared among all classes, i.e., a bounding box could be classified as any candidate class.
	 * Otherwise
	 * Bounding box are designed for specific classes, i.e., a bounding box could be classified as one certain class or not (binary classification).
	 */
	const int numLocClasses = shareLocation ? 1 : numClasses;

	size_t bboxDataSize = detectionForwardBBoxDataSize(N, C1, DataType::kFLOAT);  //for pPoints 5250 
	void* bboxDataRaw = workspace;

	pluginStatus_t status = decodeBBoxes(stream,
		locCount,
		codeType,
		varianceEncodedInTarget,
		numPredsPerClass,
		shareLocation,
		numLocClasses,
		backgroundLabelId,
		clipBBox,
		DataType::kFLOAT,
		locData,
		priorData,
		bboxDataRaw);

	ASSERT_FAILURE(status == STATUS_SUCCESS);

	/*
	 * bboxDataRaw format:
	 * [batch size, numPriors (per sample), numLocClasses, 4]
	 */
	 // float for now
	void* bboxData;
	size_t bboxPermuteSize = detectionForwardBBoxPermuteSize(shareLocation, N, C1, DataType::kFLOAT); // for pPoints 0
	void* bboxPermute = nextWorkspacePtr((int8_t*)bboxDataRaw, bboxDataSize);

	/*
	 * After permutation, bboxData format:
	 * [batch_size, numLocClasses, numPriors (per sample) (numPredsPerClass), 4]
	 * This is equivalent to swapping axis
	 */
	if (!shareLocation)
	{
		status = permuteData(stream,
			locCount,
			numLocClasses,
			numPredsPerClass,
			4,
			DataType::kFLOAT,
			false,
			bboxDataRaw,
			bboxPermute);
		ASSERT_FAILURE(status == STATUS_SUCCESS);
		bboxData = bboxPermute;
	}
	/*
	 * If shareLocation, numLocClasses = 1
	 * No need to permute data on linear memory
	 */
	else
	{
		bboxData = bboxDataRaw;
	}
	/*
	 * Conf data format
	 * [batch size, numPriors * param.numClasses, 1, 1]
	 */
	const int numScores = N * C2;
	size_t scoresSize = detectionForwardPreNMSSize(N, C2); //for pPoints 4200
	void* scores = nextWorkspacePtr((int8_t*)bboxPermute, bboxPermuteSize);
	// need a conf_scores
	/*
	 * After permutation, bboxData format:
	 * [batch_size, numClasses, numPredsPerClass, 1]
	 */
	status = permuteData(stream,
		numScores,
		numClasses,
		numPredsPerClass,
		1,
		DataType::kFLOAT,
		confSigmoid,
		confData,
		scores);
	ASSERT_FAILURE(status == STATUS_SUCCESS);

	size_t indicesSize = detectionForwardPreNMSSize(N, C2); //for pPoints 4200
	void* indices = nextWorkspacePtr((int8_t*)scores, scoresSize);

	size_t postNMSScoresSize = detectionForwardPostNMSSize(N, numClasses, topK); //for pPoints 1600
	size_t postNMSIndicesSize = detectionForwardPostNMSSize(N, numClasses, topK); //for pPoints 1600
	void* postNMSScores = nextWorkspacePtr((int8_t*)indices, indicesSize);
	void* postNMSIndices = nextWorkspacePtr((int8_t*)postNMSScores, postNMSScoresSize);

	//size_t sortingWorkspaceSize = sortScoresPerClassWorkspaceSize(N, numClasses, numPredsPerClass, FLOAT32);
	void* sortingWorkspace = nextWorkspacePtr((int8_t*)postNMSIndices, postNMSIndicesSize);
	// Sort the scores so that the following NMS could be applied.
	status = sortScoresPerClass(stream,
		N,
		numClasses,
		numPredsPerClass,
		backgroundLabelId,
		confidenceThreshold,
		DataType::kFLOAT,
		scores,
		indices,
		sortingWorkspace);
	ASSERT_FAILURE(status == STATUS_SUCCESS);

	// NMS
	status = allClassNMS_m(stream,
		N,
		numClasses,
		numPredsPerClass,
		topK,
		nmsThreshold,
		shareLocation,
		isNormalized,
		DataType::kFLOAT,
		DataType::kFLOAT,
		bboxData,
		scores,
		indices,
		postNMSScores,
		postNMSIndices,
		false);
	ASSERT_FAILURE(status == STATUS_SUCCESS);

	// Sort the bounding boxes after NMS using scores
	status = sortScoresPerImage(stream,
		N,
		numClasses * topK,
		DataType::kFLOAT,
		postNMSScores,
		postNMSIndices,
		scores,
		indices,
		sortingWorkspace);
	ASSERT_FAILURE(status == STATUS_SUCCESS);

	// Gather data from the sorted bounding boxes after NMS
	status = gatherTopDetections(stream,
		shareLocation,
		N,
		numPredsPerClass,
		numClasses,
		topK,
		keepTopK,
		DataType::kFLOAT,
		DataType::kFLOAT,
		indices,
		scores,
		bboxData,
		keepCount,
		topDetections);
	ASSERT_FAILURE(status == STATUS_SUCCESS);

	return STATUS_SUCCESS;
}
