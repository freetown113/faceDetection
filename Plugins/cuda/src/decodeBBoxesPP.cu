#include "hip/hip_runtime.h"
#include <vector>
#include "kernel.h"

template <typename T_BBOX, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
__global__ void decodeBBoxes_kernelPP(
	const int nthreads,
	const CodeTypeSSD code_type,
	const bool variance_encoded_in_target,
	const int num_priors,
	const bool share_location,
	const int num_loc_classes,
	const int background_label_id,
	const bool clip_bbox,
	const T_BBOX* loc_data,
	const T_BBOX* prior_data,
	T_BBOX* bbox_data)
{
	for (int index = blockIdx.x * nthds_per_cta + threadIdx.x;
		index < nthreads;
		index += nthds_per_cta * gridDim.x)
	{
		// Particular points coordinate index {0, 1, 2, 3, 4}
		const int i = index % 10;
		// Particular points set class index
		const int c = (index / 10) % num_loc_classes;
		// Particular points set id corresponding to the particular points
		const int d = (index / 10 / num_loc_classes) % num_priors;
		// If Particular points set was not shared among all the classes and the Particular points set is corresponding to the background class
		if (!share_location && c == background_label_id)
		{
			// Ignore background class if not share_location.
			return;
		}
		// Index to the right anchor box corresponding to the current Particular points 
		const int pi = d * 4;
		// Index to the right variances corresponding to the current Particular points	 
		const int vi = pi + num_priors * 4;

		// Encoding method: CodeTypeSSD::CORNER
		//if (code_type == PriorBoxParameter_CodeType_CORNER){
		if (code_type == CodeTypeSSD::CORNER)
		{
			// Do not want to use variances to adjust the bounding box decoding
			if (variance_encoded_in_target)
			{
				// variance is encoded in target, we simply need to add the offset
				// predictions.
				// prior_data[pi + i]: prior box coordinates corresponding to the current bounding box coordinate
				bbox_data[index] = prior_data[pi + i] + loc_data[index];
			}
			else
			{
				// variance is encoded in bbox, we need to scale the offset accordingly.
				// prior_data[vi + i]: variance corresponding to the current bounding box coordinate
				bbox_data[index] = prior_data[pi + i] + loc_data[index] * prior_data[vi + i];
			}
			//} else if (code_type == PriorBoxParameter_CodeType_CENTER_SIZE) {
		}
		// Encoding method: CodeTypeSSD::CENTER_SIZE
		else if (code_type == CodeTypeSSD::CENTER_SIZE)
		{
			// Get prior box coordinates
			const T_BBOX p_xmin = prior_data[pi];
			const T_BBOX p_ymin = prior_data[pi + 1];
			const T_BBOX p_xmax = prior_data[pi + 2];
			const T_BBOX p_ymax = prior_data[pi + 3];

			// Calculate prior box center, height, and width
			/*const T_BBOX prior_width = p_xmax - p_xmin;
			const T_BBOX prior_height = p_ymax - p_ymin;
			const T_BBOX prior_center_x = (p_xmin + p_xmax) / 2.;
			const T_BBOX prior_center_y = (p_ymin + p_ymax) / 2.;*/
			//printf("index = %d, index - i = %d \n", index, index - i);
			// Get the current bounding box coordinates
			const T_BBOX loc_x1 = loc_data[index - i];
			const T_BBOX loc_y1 = loc_data[index - i + 1];
			const T_BBOX loc_x2 = loc_data[index - i + 2];
			const T_BBOX loc_y2 = loc_data[index - i + 3];
			const T_BBOX loc_x3 = loc_data[index - i + 4];
			const T_BBOX loc_y3 = loc_data[index - i + 5];
			const T_BBOX loc_x4 = loc_data[index - i + 6];
			const T_BBOX loc_y4 = loc_data[index - i + 7];
			const T_BBOX loc_x5 = loc_data[index - i + 8];
			const T_BBOX loc_y5 = loc_data[index - i + 9];
			// Declare decoded bounding box coordinates
			T_BBOX decode_bbox_center_x1, decode_bbox_center_x2, decode_bbox_center_x3, decode_bbox_center_x4, decode_bbox_center_x5;
			T_BBOX decode_bbox_center_y1, decode_bbox_center_y2, decode_bbox_center_y3, decode_bbox_center_y4, decode_bbox_center_y5;
			// Do not want to use variances to adjust the bounding box decoding
			//if (variance_encoded_in_target)
			//{
			//	// variance is encoded in target, we simply need to retore the offset
			//	// predictions.
			//	//decode_bbox_center_x1 = xmin * prior_width + prior_center_x;
			//	//decode_bbox_center_y1 = ymin * prior_height + prior_center_y;
			//	//decode_bbox_width = exp(xmax) * prior_width;
			//	//decode_bbox_height = exp(ymax) * prior_height;

			//}
			//else
			//{
				// variance is encoded in bbox, we need to scale the offset accordingly.
				//decode_bbox_center_x = prior_data[vi] * xmin * prior_width + prior_center_x;
				//decode_bbox_center_y = prior_data[vi + 1] * ymin * prior_height + prior_center_y;
				//decode_bbox_width = exp(prior_data[vi + 2] * xmax) * prior_width;
				//decode_bbox_height = exp(prior_data[vi + 3] * ymax) * prior_height;
				decode_bbox_center_x1 = p_xmin + loc_x1 * 0.1 * p_xmax;
				decode_bbox_center_y1 = p_ymin + loc_y1 * 0.1 * p_ymax;
				decode_bbox_center_x2 = p_xmin + loc_x2 * 0.1 * p_xmax;
				decode_bbox_center_y2 = p_ymin + loc_y2 * 0.1 * p_ymax;
				decode_bbox_center_x3 = p_xmin + loc_x3 * 0.1 * p_xmax;
				decode_bbox_center_y3 = p_ymin + loc_y3 * 0.1 * p_ymax;
				decode_bbox_center_x4 = p_xmin + loc_x4 * 0.1 * p_xmax;
				decode_bbox_center_y4 = p_ymin + loc_y4 * 0.1 * p_ymax;
				decode_bbox_center_x5 = p_xmin + loc_x5 * 0.1 * p_xmax;
				decode_bbox_center_y5 = p_ymin + loc_y5 * 0.1 * p_ymax;
			//}

			// Use [x_topleft, y_topleft, x_bottomright, y_bottomright] as coordinates for final decoded bounding box output
			
			switch (i)
			{
			case 0:
				bbox_data[index] = decode_bbox_center_x1;
				break;
			case 1:
				bbox_data[index] = decode_bbox_center_y1;
				break;
			case 2:
				bbox_data[index] = decode_bbox_center_x2; //decode_bbox_center_x + decode_bbox_width / 2.;
				break;
			case 3:
				bbox_data[index] = decode_bbox_center_y2; //decode_bbox_center_y + decode_bbox_height / 2.;
				break;
			case 4:
				bbox_data[index] = decode_bbox_center_x3; //decode_bbox_center_y + decode_bbox_height / 2.;
				break;
			case 5:
				bbox_data[index] = decode_bbox_center_y3;
				break;
			case 6:
				bbox_data[index] = decode_bbox_center_x4;
				break;
			case 7:
				bbox_data[index] = decode_bbox_center_y4; //decode_bbox_center_x + decode_bbox_width / 2.;
				break;
			case 8:
				bbox_data[index] = decode_bbox_center_x5; //decode_bbox_center_y + decode_bbox_height / 2.;
				break;
			case 9:
				bbox_data[index] = decode_bbox_center_y5; //decode_bbox_center_y + decode_bbox_height / 2.;
				break;
			}
			//} else if (code_type == PriorBoxParameter_CodeType_CORNER_SIZE) {
		}
		// Encoding method: CodeTypeSSD::CORNER_SIZE
		else if (code_type == CodeTypeSSD::CORNER_SIZE)
		{
			// Get prior box coordinates
			const T_BBOX p_xmin = prior_data[pi];
			const T_BBOX p_ymin = prior_data[pi + 1];
			const T_BBOX p_xmax = prior_data[pi + 2];
			const T_BBOX p_ymax = prior_data[pi + 3];
			// Get prior box width and height
			const T_BBOX prior_width = p_xmax - p_xmin;
			const T_BBOX prior_height = p_ymax - p_ymin;
			T_BBOX p_size;
			if (i == 0 || i == 2)
			{
				p_size = prior_width;
			}
			else
			{
				p_size = prior_height;
			}
			// Do not want to use variances to adjust the bounding box decoding
			if (variance_encoded_in_target)
			{
				// variance is encoded in target, we simply need to add the offset
				// predictions.
				bbox_data[index] = prior_data[pi + i] + loc_data[index] * p_size;
			}
			else
			{
				// variance is encoded in bbox, we need to scale the offset accordingly.
				bbox_data[index] = prior_data[pi + i] + loc_data[index] * prior_data[vi + i] * p_size;
			}
		}
		// Exactly the same to CodeTypeSSD::CENTER_SIZE with using variance to adjust the bounding box decoding 
		else if (code_type == CodeTypeSSD::TF_CENTER)
		{
			const T_BBOX pXmin = prior_data[pi];
			const T_BBOX pYmin = prior_data[pi + 1];
			const T_BBOX pXmax = prior_data[pi + 2];
			const T_BBOX pYmax = prior_data[pi + 3];
			const T_BBOX priorWidth = pXmax - pXmin;
			const T_BBOX priorHeight = pYmax - pYmin;
			const T_BBOX priorCenterX = (pXmin + pXmax) / 2.;
			const T_BBOX priorCenterY = (pYmin + pYmax) / 2.;

			const T_BBOX ymin = loc_data[index - i];
			const T_BBOX xmin = loc_data[index - i + 1];
			const T_BBOX ymax = loc_data[index - i + 2];
			const T_BBOX xmax = loc_data[index - i + 3];

			T_BBOX bboxCenterX, bboxCenterY;
			T_BBOX bboxWidth, bboxHeight;

			bboxCenterX = prior_data[vi] * xmin * priorWidth + priorCenterX;
			bboxCenterY = prior_data[vi + 1] * ymin * priorHeight + priorCenterY;
			bboxWidth = exp(prior_data[vi + 2] * xmax) * priorWidth;
			bboxHeight = exp(prior_data[vi + 3] * ymax) * priorHeight;

			switch (i)
			{
			case 0:
				bbox_data[index] = bboxCenterX - bboxWidth / 2.;
				break;
			case 1:
				bbox_data[index] = bboxCenterY - bboxHeight / 2.;
				break;
			case 2:
				bbox_data[index] = bboxCenterX + bboxWidth / 2.;
				break;
			case 3:
				bbox_data[index] = bboxCenterY + bboxHeight / 2.;
				break;
			}
		}
		else
		{
			// Unknown code type.
			assert("Unknown Box decode code type");
		}
		// Clip bounding box or not
		if (clip_bbox)
		{
			bbox_data[index] = max(min(bbox_data[index], T_BBOX(1.)), T_BBOX(0.));
		}
	}
	//for (int index = blockIdx.x * nthds_per_cta + threadIdx.x;
	//	index < nthreads;
	//	index += nthds_per_cta * gridDim.x)
	//{
	//	printf("index %d: %f \n", index, bbox_data[index]);
	//}
}

template <typename T_BBOX>
pluginStatus_t decodeBBoxes_gpuPP(
	hipStream_t stream,
	const int nthreads,
	const CodeTypeSSD code_type,
	const bool variance_encoded_in_target,
	const int num_priors,
	const bool share_location,
	const int num_loc_classes,
	const int background_label_id,
	const bool clip_bbox,
	const void* loc_data,
	const void* prior_data,
	void* bbox_data)
{
	const int BS = 512;
	const int GS = (nthreads + BS - 1) / BS;
	decodeBBoxes_kernelPP<T_BBOX, BS> << <GS, BS, 0, stream >> > (nthreads, code_type, variance_encoded_in_target,
		num_priors, share_location, num_loc_classes,
		background_label_id, clip_bbox,
		(const T_BBOX*)loc_data, (const T_BBOX*)prior_data,
		(T_BBOX*)bbox_data);
	CSC(hipGetLastError(), STATUS_FAILURE);
	return STATUS_SUCCESS;
}

// decodeBBoxes LAUNCH CONFIG
typedef pluginStatus_t(*dbbFunc)(hipStream_t,
	const int,
	const CodeTypeSSD,
	const bool,
	const int,
	const bool,
	const int,
	const int,
	const bool,
	const void*,
	const void*,
	void*);

struct dbbLaunchConfig
{
	DataType t_bbox;
	dbbFunc function;

	dbbLaunchConfig(DataType t_bbox)
		: t_bbox(t_bbox)
	{
	}
	dbbLaunchConfig(DataType t_bbox, dbbFunc function)
		: t_bbox(t_bbox)
		, function(function)
	{
	}
	bool operator==(const dbbLaunchConfig& other)
	{
		return t_bbox == other.t_bbox;
	}
};

static std::vector<dbbLaunchConfig> dbbFuncVec;

bool decodeBBoxesInitPP()
{
	dbbFuncVec.push_back(dbbLaunchConfig(DataType::kFLOAT, decodeBBoxes_gpuPP<float>));
	return true;
}

static bool initialized = decodeBBoxesInitPP();

pluginStatus_t decodeBBoxesPP(
	hipStream_t stream,
	const int nthreads,
	const CodeTypeSSD code_type,
	const bool variance_encoded_in_target,
	const int num_priors,
	const bool share_location,
	const int num_loc_classes,
	const int background_label_id,
	const bool clip_bbox,
	const DataType DT_BBOX,
	const void* loc_data,
	const void* prior_data,
	void* bbox_data)
{
	dbbLaunchConfig lc = dbbLaunchConfig(DT_BBOX);
	for (unsigned i = 0; i < dbbFuncVec.size(); ++i)
	{
		if (lc == dbbFuncVec[i])
		{
			DEBUG_PRINTF("decodeBBox kernel %d\n", i);
			return dbbFuncVec[i].function(stream,
				nthreads,
				code_type,
				variance_encoded_in_target,
				num_priors,
				share_location,
				num_loc_classes,
				background_label_id,
				clip_bbox,
				loc_data,
				prior_data,
				bbox_data);
		}
	}
	return STATUS_BAD_PARAM;
}